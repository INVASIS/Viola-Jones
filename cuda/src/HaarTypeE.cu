
#include <hip/hip_runtime.h>
__device__
int rectanglesSum(int** integralImage, int x, int y, int w, int h)
{
    int A = x > 0 && y > 0 ? integralImage[x - 1][y - 1] : 0;
    int B = x + w > 0 && y > 0 ? integralImage[x + w - 1][y - 1] : 0;
    int C = x > 0 && y + h > 0 ? integralImage[x - 1][y + h - 1] : 0;
    int D = x + w > 0 && y + h > 0 ? integralImage[x + w - 1][y + h - 1] : 0;

    return A + D - B - C;
}

extern "C"
__global__ void haar_type_E(int** integralImage, int* allRectangles, int numRectangles, float coeff, int* haarFeatures)
{
    // Get an "unique id" of the thread that correspond to one pixel
    const unsigned int tidX = blockIdx.x * blockDim.x + threadIdx.x;

    if (tidX < numRectangles)
    {

        int x = (int) (allRectangles[tidX * 4] * coeff);
        int y = (int) (allRectangles[tidX * 4 + 1] * coeff);
        int w = (int) (allRectangles[tidX * 4 + 2] * coeff);
        int h = (int) (allRectangles[tidX * 4 + 3] * coeff);

        int mid_w = w / 2;
        int mid_h = h / 2;

        int r1 = rectanglesSum(integralImage, x, y, mid_w, mid_h);
        int r2 = rectanglesSum(integralImage, x + mid_w, y, mid_w, mid_h);
        int r3 = rectanglesSum(integralImage, x, y + mid_h, mid_w, mid_h);
        int r4 = rectanglesSum(integralImage, x + mid_w, y + mid_h, mid_w, mid_h);

        haarFeatures[tidX] = r1 - r2 - r3 + r4;
    }

    __syncthreads();
}
