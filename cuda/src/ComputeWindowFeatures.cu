
#include <hip/hip_runtime.h>
__device__
int rectanglesSum(int** integralImage, int x, int y, int w, int h)
{
	int A = x > 0 && y > 0 ? integralImage[x - 1][y - 1] : 0;
	int B = x + w > 0 && y > 0 ? integralImage[x + w - 1][y - 1] : 0;
	int C = x > 0 && y + h > 0 ? integralImage[x - 1][y + h - 1] : 0;
	int D = x + w > 0 && y + h > 0 ? integralImage[x + w - 1][y + h - 1] : 0;

	return A + D - B - C;
}

extern "C"
	__global__
void computeWindowFeatures(int** integralImage, int* features, int totalNumFeatures, float* window, int* haarFeatures)
{
	// Get an "unique id" of the thread
	const unsigned int tidX = blockIdx.x * blockDim.x + threadIdx.x;

	if (tidX < totalNumFeatures)
	{
		int type = features[threadIdx.x * 5];
		int x = features[threadIdx.x * 5 + 1] + (int)window[blockIdx.x * 2];
		int y = features[threadIdx.x * 5 + 2] + (int)window[blockIdx.x * 2 + 1];
		int w = (int) (((float) (features[threadIdx.x * 5 + 3])) * window[blockIdx.x * 2 + 2]);
		int h = (int) (((float) (features[threadIdx.x * 5 + 4])) * window[blockIdx.x * 2 + 2]);

		if (type == 1)
		{
			int mid = w / 2;
			int r1 = rectanglesSum(integralImage, x, y, mid, h);
			int r2 = rectanglesSum(integralImage, x + mid, y, mid, h);
			haarFeatures[tidX] = r1 - r2;
		}
		else if (type == 2)
		{
			int mid = w / 3;

			int r1 = rectanglesSum(integralImage, x, y, mid, h);
			int r2 = rectanglesSum(integralImage, x + mid, y, mid, h);
			int r3 = rectanglesSum(integralImage, x + 2 * mid, y, mid, h);

			haarFeatures[tidX] = r1 - r2 + r3;
		}
		else if (type == 3)
		{
			int mid = h / 2;
			int r1 = rectanglesSum(integralImage, x, y, w, mid);
			int r2 = rectanglesSum(integralImage, x, y + mid, w, mid);
			haarFeatures[tidX] = r2 - r1;
		}
		else if (type == 4)
		{
			int mid = h / 3;

			int r1 = rectanglesSum(integralImage, x, y, w, mid);
			int r2 = rectanglesSum(integralImage, x, y + mid, w, mid);
			int r3 = rectanglesSum(integralImage, x, y + 2 * mid, w, mid);

			haarFeatures[tidX] = r1 - r2 + r3;
		}
		else if (type == 5)
		{
			int mid_w = w / 2;
			int mid_h = h / 2;

			int r1 = rectanglesSum(integralImage, x, y, mid_w, mid_h);
			int r2 = rectanglesSum(integralImage, x + mid_w, y, mid_w, mid_h);
			int r3 = rectanglesSum(integralImage, x, y + mid_h, mid_w, mid_h);
			int r4 = rectanglesSum(integralImage, x + mid_w, y + mid_h, mid_w, mid_h);

			haarFeatures[tidX] = r1 - r2 - r3 + r4;
		}
	}

	__syncthreads();
}
